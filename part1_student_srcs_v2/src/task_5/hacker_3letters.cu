
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__constant__ const unsigned int s_table[] = {
7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22,  7, 12, 17, 22 ,
5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20,  5,  9, 14, 20 ,
4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23,  4, 11, 16, 23 ,
6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21,  6, 10, 15, 21 };

__constant__ const unsigned int k_table[] = {
0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee ,
0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501 ,
0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be ,
0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821 ,
0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa ,
0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8 ,
0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed ,
0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a ,
0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c ,
0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70 ,
0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05 ,
0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665 ,
0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039 ,
0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1 ,
0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1 ,
0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };



const int digests_3letters[] = {
0xbc519d9f, 0xca21ef70, 0x07f3145c, 0xd8290a98, 
0xe9475db1, 0x63ee3198, 0xf37cf4e3, 0x9a8e47d4, 
0x76dab734, 0x98d2214b, 0x047d30ef, 0xc52d15d8, 
0x1c77e47f, 0xeb228a00, 0x7df43d76, 0xaac6e219, 
0x06a3c1b8, 0x7e246791, 0xdaf00335, 0x23576cba, 
0x2285c2d4, 0x1d539374, 0x0a147705, 0x6439d01e, 
0x4b90f674, 0x5fdedeb8, 0xc2dcad8e, 0xce312a0a, 
0xb2211727, 0x6962b154, 0xf5c3146e, 0xe3f86d5a, 
0xb860bfb3, 0xb2aeeb51, 0xa3018b76, 0x2ff32e2e, 
0x747d75a4, 0x483bff19, 0x59902ee9, 0x48750e6f 
};


const int digests_4letters[] = {
0x03c82e91, 0xe449ceb2, 0x8d0641a5, 0x70b55a49, 
0xe26f4639, 0x342a062b, 0x3c9fe0cf, 0x6848c2c8, 
0xd8235545, 0xb71a8a6a, 0x0832d3c7, 0xe71902fe, 
0x8fec9be7, 0xa1f82f9e, 0xf7382938, 0xcc585413, 
0x283b084b, 0xa1858b5a, 0x455602e6, 0xe39cd3da, 
0xe8ffa0f2, 0x4fd4c83e, 0x24b6e42b, 0xde7df4b0, 
0xc9418d85, 0xfab897e3, 0x6d04bb34, 0x76f25580, 
0x4a6bcc95, 0xaddeabc5, 0xa8734bc7, 0xd90072ba, 
0xa4ae137c, 0xde6d6e7d, 0xc0d262fd, 0xa4b25306, 
0xf567183e, 0x4530e8ae, 0x35be5f77, 0xe13c6a5e 
};

//opencl, vulkan, mcaaca, bigbox, 
const int digests_6letters[] = {
0xb15d1993, 0x8f7873ad, 0x509361e9, 0x87887d3a, 
0x9ef17384, 0x21e22965, 0x759e4557, 0x85f4cbb4, 
0x8770cff1, 0x1ebae88f, 0x0e30356d, 0x5bc404db, 
0xe6679a67, 0x69bea3a1, 0x7c534292, 0x1dec5ee1 
};
	
#define MAX_DG (10)
#define COMB 26*26*26*26
#define Xnum 26
#define Ynum 26
#define Lnum 3  // number of letters

__device__ void md5(char* message,int length, int* digest) 		// Simplified for max. 8 letters
{
	 unsigned int a0 = 0x67452301;
	unsigned int b0 = 0xefcdab89; 
   unsigned int c0 = 0x98badcfe; 
   unsigned int d0 = 0x10325476; 
	unsigned int A=a0;
	unsigned int B=b0;
	unsigned int C=c0;
	unsigned int D=d0;
	unsigned int M[16]  = {0,0,0,0, 0,0,0,0, 0,0,0,0 , 0,0,0,0};
	memcpy(M,message,length);
	((char*)M)[length]=0x80;
	M[14]=length*8;
	for (int i=0;i<64;i++) 
	{
		unsigned int F = (B & C) | ((~B) & D);
		unsigned int G = (D & B) | ((~D) & C);
		unsigned int H = B ^ C ^ D;
		unsigned int I = C ^ (B | (~D));
		unsigned int tempD = D;
		D = C;
		C = B;
		unsigned int X=I;
		unsigned int g=(7*i) & 15;
		if (i < 48) { X = H; g=(3*i+5) & 15; }
		if (i < 32) { X = G; g=(5*i+1) & 15; }
		if (i < 16) { X = F; g=i; }

		unsigned int tmp = A+X+k_table[i]+M[g];
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
		A = tempD;
	}
   digest[0] = a0 + A;
   digest[1] = b0 + B;
   digest[2] = c0 + C;
   digest[3] = d0 + D;
   /* unsigned int a0 = 0x67452301;
	unsigned int b0 = 0xefcdab89; 
    unsigned int c0 = 0x98badcfe; 
    unsigned int d0 = 0x10325476; 
	unsigned int A=a0;
	unsigned int B=b0;
	unsigned int C=c0;
	unsigned int D=d0;
	unsigned int M[16]  = {0,0,0,0, 0,0,0,0, 0,0,0,0 , 0,0,0,0};
	memcpy(M,message,length);
	((char*)M)[length]=0x80;
	M[14]=length*8;
	
	#pragma unroll
	for(int i = 0; i < 16; i++){
		unsigned int X = (B & C) | ((~B) & D);
		unsigned int g = i;
		unsigned int tmp = A+X+k_table[i]+M[g];
		A = D;
		D = C;
		C = B;	
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
	}

	#pragma unroll
	for(int i = 16; i < 32; i++){
		unsigned int X = (D & B) | ((~D) & C);
		unsigned int g = (5*i+1) & 15;;
		unsigned int tmp = A+X+k_table[i]+M[g];
		A = D;
		D = C;
		C = B;
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
	}


	#pragma unroll
	for(int i = 32; i < 48; i++){
		unsigned int X = B ^ C ^ D;
		unsigned int g = (3*i+5) & 15;
		unsigned int tmp = A+X+k_table[i]+M[g];
		A = D;
		D = C;
		C = B;
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
	}

	#pragma unroll
	for(int i = 48; i < 64; i++){
		unsigned int X = C ^ (B | (~D));
		unsigned int g = (7*i) & 15;
		unsigned int tmp = A+X+k_table[i]+M[g];		
		A = D;
		D = C;
		C = B;
		B = B + ((tmp << s_table[i]) | ((tmp & 0xffffffff) >> (32-s_table[i])));
	}

   digest[0] = a0 + A;
   digest[1] = b0 + B;
   digest[2] = c0 + C;
   digest[3] = d0 + D;*/
}


__global__ void check_password(char *digests_GPU, const int *digests, int num_digests)
{
	int dg[4];
	char passwd_temp[Lnum+1];
 	passwd_temp[0] = 'a' + blockIdx.x;
 	//passwd_temp[1] = 'a' + blockIdx.y;
 	passwd_temp[1] = 'a' + threadIdx.x;
 	passwd_temp[2] = 'a' + threadIdx.y;

 	passwd_temp[Lnum] = 0;
	// char to int
	md5(passwd_temp,Lnum,dg);
	for (int i=0;i< num_digests; i++)
	{	
		//&& ( dg[3] == digests[i*4+3] )
		if (( dg[0] == digests[i*4] ) && ( dg[1] == digests[i*4+1] ) && ( dg[2] == digests[i*4+2] ) ) {
			memcpy(&digests_GPU[i * (Lnum+1)], passwd_temp, Lnum+1);
		}
	}
}

// totally 26*26*26*26 = 456976 combinations 
// for 26*26*26 : we have 26 * 26 * 26
int main(int argc, char** args) 
{
	char passwd[MAX_DG][Lnum+1]; 
	char* digests_GPU;

	// allocate memory space in GPU for found passwords
	hipMalloc((char**)&digests_GPU, MAX_DG*(Lnum+1));

	hipMemset(digests_GPU, 'a', MAX_DG*(Lnum+1)*sizeof(char));

	dim3 threadsPerBlock(Xnum,Ynum);
	dim3 numBlocks(Xnum,1);

	check_password <<<numBlocks, threadsPerBlock>>> (digests_GPU, digests_3letters, MAX_DG);
		
	hipMemcpy(passwd, digests_GPU, MAX_DG*(Lnum+1), hipMemcpyDeviceToHost);

	for (int i = 0; i < MAX_DG; i++)
	{
		printf("%i: %s\n", i, passwd[i]);
		
	}	
}

